#include "hip/hip_runtime.h"
/* =========================================================================
   Copyright (c) 2010-2013, Institute for Microelectronics,
                            Institute for Analysis and Scientific Computing,
                            TU Wien.
   Portions of this software are copyright by UChicago Argonne, LLC.

                            -----------------
                  ViennaCL - The Vienna Computing Library
                            -----------------

   Project Head:    Karl Rupp                   rupp@iue.tuwien.ac.at

   (A list of authors and contributors can be found in the PDF manual)

   License:         MIT (X11), see file LICENSE in the base directory
============================================================================= */

// include necessary system headers
#include <iostream>

#include "viennacl.hpp"
#include "helper.hpp"

//include basic scalar and vector types of ViennaCL
#include "viennacl/scalar.hpp"
#include "viennacl/vector.hpp"

//include the generic inner product functions of ViennaCL
#include "viennacl/linalg/inner_prod.hpp"

//include the generic norm functions of ViennaCL
#include "viennacl/linalg/norm_1.hpp"
#include "viennacl/linalg/norm_2.hpp"
#include "viennacl/linalg/norm_inf.hpp"


#ifdef VIENNACL_WITH_CUDA


// IxAMAX

ViennaCLStatus ViennaCLCUDAiSamax(ViennaCLCUDABackend backend, int n,
                                  size_t *index,
                                  float *x, int offx, int incx)
{
  viennacl::vector_base<float> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);

  *index = viennacl::linalg::index_norm_inf(v1);
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDAiDamax(ViennaCLCUDABackend backend, int n,
                                  size_t *index,
                                  double *x, int offx, int incx)
{
  viennacl::vector_base<double> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);

  *index = viennacl::linalg::index_norm_inf(v1);
  return ViennaCLSuccess;
}



// xASUM

ViennaCLStatus ViennaCLCUDASasum(ViennaCLCUDABackend backend, int n,
                                 float *alpha,
                                 float *x, int offx, int incx)
{
  viennacl::vector_base<float> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);

  *alpha = viennacl::linalg::norm_1(v1);
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADasum(ViennaCLCUDABackend backend, int n,
                                 double *alpha,
                                 double *x, int offx, int incx)
{
  viennacl::vector_base<double> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);

  *alpha = viennacl::linalg::norm_1(v1);
  return ViennaCLSuccess;
}


// xAXPY

ViennaCLStatus ViennaCLCUDASaxpy(ViennaCLCUDABackend backend, int n,
                                 float alpha,
                                 float *x, int offx, int incx,
                                 float *y, int offy, int incy)
{
  viennacl::vector_base<float> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);
  viennacl::vector_base<float> v2(y, n, viennacl::CUDA_MEMORY, offy, incy);

  v2 += alpha * v1;
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADaxpy(ViennaCLCUDABackend backend, int n,
                                 double alpha,
                                 double *x, int offx, int incx,
                                 double *y, int offy, int incy)
{
  viennacl::vector_base<double> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);
  viennacl::vector_base<double> v2(y, n, viennacl::CUDA_MEMORY, offy, incy);

  v2 += alpha * v1;
  return ViennaCLSuccess;
}


// xCOPY

ViennaCLStatus ViennaCLCUDAScopy(ViennaCLCUDABackend backend, int n,
                                 float *x, int offx, int incx,
                                 float *y, int offy, int incy)
{
  viennacl::vector_base<float> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);
  viennacl::vector_base<float> v2(y, n, viennacl::CUDA_MEMORY, offy, incy);

  v2 = v1;
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADcopy(ViennaCLCUDABackend backend, int n,
                                 double *x, int offx, int incx,
                                 double *y, int offy, int incy)
{
  viennacl::vector_base<double> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);
  viennacl::vector_base<double> v2(y, n, viennacl::CUDA_MEMORY, offy, incy);

  v2 = v1;
  return ViennaCLSuccess;
}

// xDOT

ViennaCLStatus ViennaCLCUDASdot(ViennaCLCUDABackend backend, int n,
                                float *alpha,
                                float *x, int offx, int incx,
                                float *y, int offy, int incy)
{
  viennacl::vector_base<float> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);
  viennacl::vector_base<float> v2(y, n, viennacl::CUDA_MEMORY, offy, incy);

  *alpha = viennacl::linalg::inner_prod(v1, v2);
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADdot(ViennaCLCUDABackend backend, int n,
                                double *alpha,
                                double *x, int offx, int incx,
                                double *y, int offy, int incy)
{
  viennacl::vector_base<double> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);
  viennacl::vector_base<double> v2(y, n, viennacl::CUDA_MEMORY, offy, incy);

  *alpha = viennacl::linalg::inner_prod(v1, v2);
  return ViennaCLSuccess;
}

// xNRM2

ViennaCLStatus ViennaCLCUDASnrm2(ViennaCLCUDABackend backend, int n,
                                 float *alpha,
                                 float *x, int offx, int incx)
{
  viennacl::vector_base<float> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);

  *alpha = viennacl::linalg::norm_2(v1);
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADnrm2(ViennaCLCUDABackend backend, int n,
                                 double *alpha,
                                 double *x, int offx, int incx)
{
  viennacl::vector_base<double> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);

  *alpha = viennacl::linalg::norm_2(v1);
  return ViennaCLSuccess;
}



// xROT

ViennaCLStatus ViennaCLCUDASrot(ViennaCLCUDABackend backend, int n,
                                float *x, int offx, int incx,
                                float *y, int offy, int incy,
                                float c, float s)
{
  viennacl::vector_base<float> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);
  viennacl::vector_base<float> v2(y, n, viennacl::CUDA_MEMORY, offy, incy);

  viennacl::linalg::plane_rotation(v1, v2, c, s);
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADrot(ViennaCLCUDABackend backend, int n,
                                double *x, int offx, int incx,
                                double *y, int offy, int incy,
                                double c, double s)
{
  viennacl::vector_base<double> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);
  viennacl::vector_base<double> v2(y, n, viennacl::CUDA_MEMORY, offy, incy);

  viennacl::linalg::plane_rotation(v1, v2, c, s);
  return ViennaCLSuccess;
}



// xSCAL

ViennaCLStatus ViennaCLCUDASscal(ViennaCLCUDABackend backend, int n,
                                 float alpha,
                                 float *x, int offx, int incx)
{
  viennacl::vector_base<float> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);

  v1 *= alpha;
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADscal(ViennaCLCUDABackend backend, int n,
                                 double alpha,
                                 double *x, int offx, int incx)
{
  viennacl::vector_base<double> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);

  v1 *= alpha;
  return ViennaCLSuccess;
}


// xSWAP

ViennaCLStatus ViennaCLCUDASswap(ViennaCLCUDABackend backend, int n,
                                 float *x, int offx, int incx,
                                 float *y, int offy, int incy)
{
  viennacl::vector_base<float> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);
  viennacl::vector_base<float> v2(y, n, viennacl::CUDA_MEMORY, offy, incy);

  viennacl::swap(v1, v2);
  return ViennaCLSuccess;
}

ViennaCLStatus ViennaCLCUDADswap(ViennaCLCUDABackend backend, int n,
                                 double *x, int offx, int incx,
                                 double *y, int offy, int incy)
{
  viennacl::vector_base<double> v1(x, n, viennacl::CUDA_MEMORY, offx, incx);
  viennacl::vector_base<double> v2(y, n, viennacl::CUDA_MEMORY, offy, incy);

  viennacl::swap(v1, v2);
  return ViennaCLSuccess;
}
#endif


